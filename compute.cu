#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>

__global__ void parallel_compute(vector3 *hPos, vector3 *accels, double *mass)
{
    int column = (blockDim.x * blockIdx.x) + threadIdx.x;
    int row = (blockDim.y * blockIdx.y) + threadIdx.y;
    int ind = (NUMENTITIES * row) + column;
    int i = row;
    int j = column;
    if (ind < NUMENTITIES * NUMENTITIES)
    {
        if (i == j){
        	FILL_VECTOR(accels[ind], 0, 0, 0);
        }
        else{
            vector3 distance;
            for (int k = 0; k < 3; k++){
                distance[k] = hPos[i][k] - hPos[j][k];
            }
            double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
            double magnitude = sqrt(magnitude_sq);
            double accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
            FILL_VECTOR(accels[ind], accelmag * distance[0] / magnitude, accelmag * distance[1] / magnitude, accelmag * distance[2] / magnitude);
        }
    }
}

__global__ void parallel_sum(vector3 *accels, vector3 *accel_sum, vector3 *hPos, vector3 *hVel)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int i = row;
    if (i < NUMENTITIES){
    	FILL_VECTOR(accel_sum[i], 0, 0, 0);
        for (int j = 0; j < NUMENTITIES; j++){
            for (int k = 0; k < 3; k++){
                accel_sum[i][k] += accels[(i * NUMENTITIES) + j][k];
            }
        }
        for (int k = 0; k < 3; k++){
            hVel[i][k] += accel_sum[i][k] * INTERVAL;
            hPos[i][k] = hVel[i][k] * INTERVAL;
        }
    }
}

// compute: Updates the positions and locations of the objects in the system based on gravity.
// Parameters: None
// Returns: None
// Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute()
{
    vector3 *dev_hPos, *dev_hVel, *dev_acc, *dev_sum;
    double *dev_mass;
    int blocksD = ceilf(NUMENTITIES / 16.0f);
    int threadsD = ceilf(NUMENTITIES / (float)blocksD);
    dim3 gridDim(blocksD, blocksD, 1);
    dim3 blockDim(threadsD, threadsD, 1);
    hipMalloc(&dev_hPos,sizeof(vector3) * NUMENTITIES);
	hipMemcpy(dev_hPos,hPos,sizeof(vector3) * NUMENTITIES,hipMemcpyHostToDevice);
    hipMalloc(&dev_hVel,sizeof(vector3) * NUMENTITIES);
	hipMemcpy(dev_hVel,hVel,sizeof(vector3) * NUMENTITIES,hipMemcpyHostToDevice);
    hipMalloc(&dev_acc,sizeof(vector3) * NUMENTITIES);
    hipMalloc(&dev_sum,sizeof(vector3) * NUMENTITIES);
    hipMalloc(&dev_mass,sizeof(double) * NUMENTITIES);
    hipMemcpy(dev_mass,mass,sizeof(double) * NUMENTITIES,hipMemcpyHostToDevice);
    Parallel_Computation<<<gridDim, blockDim>>>(dev_hPos, dev_acc, dev_mass);
    hipDeviceSynchronize();
    Parallel_Sum<<<gridDim.x, blockDim.x>>>(dev_acc, dev_sum, dev_hPos, dev_hVel);
    hipDeviceSynchronize();
    hipMemcpy(hPos,dev_hPos,sizeof(vector3) * NUMENTITIES,hipMemcpyDeviceToHost);
    hipMemcpy(hVel,dev_hVel,sizeof(vector3) * NUMENTITIES,hipMemcpyDeviceToHost);
    hipFree(dev_hPos);
    hipFree(dev_hVel);
    hipFree(dev_mass);
    hipFree(dev_acc);
}
